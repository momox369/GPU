#include "hip/hip_runtime.h"

#include "common.h"

#include "timer.h"

#define BLOCK_DIM 1024

__global__ void reduce_kernel(float* input, float* sum, unsigned int N) {
    
    //Declaring a shared memory array to store the input
    __shared__ float input_s[BLOCK_DIM];
    unsigned int i = threadIdx.x + (2 * blockIdx.x * BLOCK_DIM);

    //Initialisation
    if (i < N) {
        input_s[threadIdx.x] = input[i];
    } else {
        input_s[threadIdx.x] = 0.0f;
    }

    if (i - BLOCK_DIM < N){
        input_s[threadIdx.x] = input_s[threadIdx.x] + input[i-BLOCK_DIM];
    }
    __syncthreads();

    for (unsigned int stride = BLOCK_DIM / 2; stride > 0; stride = stride/2) {
        if (threadIdx.x >= (BLOCK_DIM - stride)) {
            input_s[threadIdx.x] += input_s[threadIdx.x - stride];
        } 
        __syncthreads(); 
    }
    if (threadIdx.x == BLOCK_DIM - 1){
        atomicAdd(sum, input_s[BLOCK_DIM - 1]);
    }
}

float reduce_gpu(float* input, unsigned int N) {

    Timer timer;

    // Allocate memory
    startTime(&timer);
    float *input_d;
    hipMalloc((void**) &input_d, N*sizeof(float));
    float *sum_d; 
    hipMalloc((void**) &sum_d, sizeof(float));
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Allocation time");

    // Copy data to GPU
    startTime(&timer);
    hipMemcpy(input_d, input, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemset(sum_d, 0, sizeof(float));
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy to GPU time");

    // Call kernel
    startTime(&timer);
    const unsigned int numThreadsPerBlock = BLOCK_DIM;
    const unsigned int numElementsPerBlock = 2*numThreadsPerBlock;
    const unsigned int numBlocks = (N + numElementsPerBlock - 1)/numElementsPerBlock;
    reduce_kernel <<< numBlocks, numThreadsPerBlock >>> (input_d, sum_d, N);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Kernel time", GREEN);

    // Copy data from GPU
    startTime(&timer);
    float sum;
    hipMemcpy(&sum, sum_d, sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy from GPU time");

    // Free memory
    startTime(&timer);
    hipFree(input_d);
    hipFree(sum_d);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Deallocation time");

    return sum;

}

