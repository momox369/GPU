#include "hip/hip_runtime.h"
#include "common.h"
#include "timer.h"

#define IN_TILE_DIM 32
#define OUT_TILE_DIM ((IN_TILE_DIM) - 2*(FILTER_RADIUS))

__constant__ float filter_c[FILTER_DIM][FILTER_DIM];

__global__ void convolution_tiled_kernel(float* input, float* output, unsigned int width, unsigned int height) {
	__shared__ float in_s[IN_TILE_DIM][IN_TILE_DIM];
    int row = threadIdx.y + blockIdx.y * OUT_TILE_DIM -FILTER_RADIUS;
    int col = threadIdx.x + blockIdx.x * OUT_TILE_DIM- FILTER_RADIUS;
	if((row >=0) && (row< height ) && (col>=0) && (col < width ) ) {
        in_s[threadIdx.y][threadIdx.x]=input[row*width + col];
    }else{
        in_s[threadIdx.y][threadIdx.x]=0.0f;
    }
	__syncthreads();
    if(threadIdx.y>=FILTER_RADIUS && threadIdx.y<IN_TILE_DIM-FILTER_RADIUS && threadIdx.x>=FILTER_RADIUS && threadIdx.x< IN_TILE_DIM-FILTER_RADIUS){
		float sum = 0.0f;
        for(int i = 0; i < FILTER_DIM; i++) {
			for(int j = 0; j < FILTER_DIM; j++) { 
				sum += filter_c[i][j] * in_s[i+threadIdx.y-FILTER_RADIUS][j+threadIdx.x-FILTER_RADIUS];
            } 
        }
        if(row < height && col < width){
			output[row*width + col] = sum;
        }
    }
}

void copyFilterToGPU(float filter[][FILTER_DIM]) {
    // Copy filter to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(filter_c), filter, FILTER_DIM*FILTER_DIM*sizeof(float));

}

void convolution_tiled_gpu(float* input_d, float* output_d, unsigned int width, unsigned int height) {

    // Call kernel

    dim3 numThreadsPerBlock(IN_TILE_DIM, IN_TILE_DIM);
    dim3 numBlocks((width + OUT_TILE_DIM - 1)/OUT_TILE_DIM, (height + OUT_TILE_DIM - 1)/OUT_TILE_DIM);
    convolution_tiled_kernel <<< numBlocks, numThreadsPerBlock >>> (input_d, output_d, width, height);
}