#include "hip/hip_runtime.h"

#include "common.h"
#include "timer.h"
#define COARSE_FACTOR 32
__global__ void histogram_private_kernel(unsigned char* image, unsigned int* bins, unsigned int width, unsigned int height) {
    __shared__  int  b_s[NUM_BINS];
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(threadIdx.x<NUM_BINS){
	b_s[threadIdx.x]=0;
    }
    __syncthreads();
    
    if(i< width*height) {
        unsigned char b = image[i];
        atomicAdd(&b_s[b], 1);
    }
    __syncthreads();
    if(threadIdx.x<NUM_BINS) {
        atomicAdd(&bins[threadIdx.x], b_s[threadIdx.x]);
    }

}

void histogram_gpu_private(unsigned char* image_d, unsigned int* bins_d, unsigned int width, unsigned int height) {

    unsigned int numThreadsPerBlock=1024;
    unsigned int numBlocks=(width*height+numThreadsPerBlock-1)/numThreadsPerBlock;
    histogram_private_kernel<<<numBlocks, numThreadsPerBlock>>>(image_d, bins_d,width,height);
}

__global__ void histogram_private_coarse_kernel(unsigned char* image, unsigned int* bins, unsigned int width, unsigned int height) {

    __shared__ unsigned int bins_s [NUM_BINS];
    unsigned int idx = blockIdx.x * blockDim.x * COARSE_FACTOR + threadIdx.x;

    if ( threadIdx.x < NUM_BINS ) 
        bins_s[ threadIdx.x ] = 0;
    
    __syncthreads();

    for(int i = 0; i < COARSE_FACTOR; ++i) 
        if (i * blockDim.x + idx < width * height)
            atomicAdd(&bins_s[image[i * blockDim.x + idx ]], 1);

    __syncthreads();

    if (threadIdx.x < NUM_BINS && bins_s[threadIdx.x] > 0) 
        atomicAdd(&bins[threadIdx.x], bins_s[threadIdx.x]);
}

void histogram_gpu_private_coarse(unsigned char* image_d, unsigned int* bins_d, unsigned int width, unsigned int height) {

    unsigned int numThreadsPerBlock=1024;
    unsigned int numBlocks=(width*height+numThreadsPerBlock-1)/numThreadsPerBlock;
    histogram_private_kernel<<<numBlocks, numThreadsPerBlock>>>(image_d, bins_d,width,height);
    
}

