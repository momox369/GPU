#include "hip/hip_runtime.h"

#include "common.h"

#include "timer.h"

#define COARSENING_FACTOR 16

__global__ void histogram_private_kernel(unsigned char* image, unsigned int* bins, unsigned int width, unsigned int height) {

    // TODO
    __shared__ int bins_s[NUM_BINS];

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadIdx.x < NUM_BINS){
	    bins_s[threadIdx.x] = 0;
    }
    __syncthreads();
    
    if(i< width*height) {
        atomicAdd(&bins_s[image[i]], 1);
    }
    __syncthreads();

    if(threadIdx.x<NUM_BINS) {
        if (bins_s[threadIdx.x] > 0)
        atomicAdd(&bins[threadIdx.x], bins_s[threadIdx.x]);
    }

}

void histogram_gpu_private(unsigned char* image_d, unsigned int* bins_d, unsigned int width, unsigned int height) {

    // TODO
    int numThreadsPerBlock = 256;
    int numBlocks = (width*height + numThreadsPerBlock - 1)/(numThreadsPerBlock);
    histogram_private_kernel<<<numBlocks, numThreadsPerBlock>>>(image_d, bins_d, width, height);
}


__global__ void histogram_private_coarse_kernel(unsigned char* image, unsigned int* bins, unsigned int width, unsigned int height) {

    // TODO
    __shared__ unsigned int bins_s[NUM_BINS];

    
    int i = threadIdx.x + blockIdx.x * blockDim.x * COARSENING_FACTOR;

    //initialize bin_s to 0s
    if (threadIdx.x < NUM_BINS){
        bins_s[threadIdx.x] = 0;
    }
    __syncthreads();


    for (int k = 0; k < COARSENING_FACTOR; ++k){
        if (i + k*blockDim.x < width * height) {                
            atomicAdd(
                    &bins_s[image[i + k*blockDim.x]], 
                    1);
        }
    }
    
    __syncthreads();
    //

    //Commit the non-zero bin counts to the global copy of the histogram in parallel
    if (threadIdx.x < NUM_BINS) {
        if (bins_s[threadIdx.x] > 0)  
            atomicAdd(&bins[threadIdx.x], bins_s[threadIdx.x]);  
    }
}

void histogram_gpu_private_coarse(unsigned char* image_d, unsigned int* bins_d, unsigned int width, unsigned int height) {

    // TODO
    //Launch the grid (Note: the image has already been copied to global memory

    //Set the number of threads per block
    int numThreadsPerBlock = 256;
    int numBlocks = (width*height + numThreadsPerBlock - 1)/numThreadsPerBlock;
    histogram_private_kernel<<<numBlocks, numThreadsPerBlock>>>(image_d, bins_d, width, height);
}


