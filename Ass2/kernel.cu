#include "hip/hip_runtime.h"

#include "common.h"

#include "timer.h"

__global__ void mm_kernel(float* A, float* B, float* C, unsigned int M, unsigned int N, unsigned int K) {

    // TODO
    // rows = M and cols = K
    unsigned int row = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x*blockDim.x + threadIdx.x;

    if (row < K && col < M){
        float sum = 0.0f;
        for(unsigned int j = 0; j < N; ++j) {
            sum += A[row*N + j]*B[j*N + col];
        }
        C[row*N + col] = sum;
    }

}

void mm_gpu(float* A, float* B, float* C, unsigned int M, unsigned int N, unsigned int K) {

    Timer timer;

    // Allocate GPU memory
    startTime(&timer);

    // TODO
    float* A_d;
    float* B_d;
    float* C_d;
    hipMalloc((void**)&A_d, sizeof(float)*M*N);
    hipMalloc((void**)&B_d, sizeof(float)*N*K);
    hipMalloc((void**)&C_d, sizeof(float)*M*K);
    //

    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Allocation time");

    // Copy data to GPU
    startTime(&timer);

    // TODO
    hipMemcpy(A_d, A, sizeof(float)*M*N, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, sizeof(float)*N*K, hipMemcpyHostToDevice);
    //

    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy to GPU time");

    // Call kernel
    startTime(&timer);

    // TODO
    // C is a MxK matrix
    dim3 numThreadsPerBlock(32, 32);
    dim3 numBlocks((M + numThreadsPerBlock.x - 1)/numThreadsPerBlock.x, (K + numThreadsPerBlock.y - 1)/numThreadsPerBlock.y);
    mm_kernel <<<numBlocks, numThreadsPerBlock>>> (A_d, B_d, C_d, M, N, K);
    //

    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Kernel time", GREEN);

    // Copy data from GPU
    startTime(&timer);

    // TODO
    hipMemcpy(C, C_d, M*K*sizeof(float), hipMemcpyDeviceToHost);
    //


    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy from GPU time");

    // Free GPU memory
    startTime(&timer);

    // TODO
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
    //

    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Deallocation time");

}

