#include "hip/hip_runtime.h"

#include "common.h"

#include "timer.h"

#define IN_TILE_DIM 32
#define OUT_TILE_DIM ((IN_TILE_DIM) - 2*(FILTER_RADIUS))

__constant__ float filter_c[FILTER_DIM][FILTER_DIM];

__global__ void convolution_tiled_kernel(float* input, float* output, unsigned int width, unsigned int height) {
    //Bring the input tile to shared memory
    __shared__ float input_tile[IN_TILE_DIM][IN_TILE_DIM];
    int in_row = (blockIdx.y * OUT_TILE_DIM) + threadIdx.y; //should I add -1 here?
    int in_col = (blockIdx.x * OUT_TILE_DIM) + threadIdx.x;

    //loading
    if ((in_row >= 0) && (in_row < height ) && (in_col >= 0) && (in_col < width ) ) {
        input_tile[threadIdx.y][threadIdx.x] = input[in_row*width + in_col];
    } else {
        input_tile[threadIdx.y][threadIdx.x] = 0.0f;
    }
    __syncthreads();

    //Compute filter x input_tile
    float sum = 0.0f;
    for (int filter_row = 0; filter_row < FILTER_DIM; ++filter_row){

        for (int filter_col = 0; filter_col < FILTER_DIM; ++filter_col){

            int out_row = in_row + filter_row - FILTER_RADIUS;
            int out_col = in_col + filter_col - FILTER_RADIUS;

            if ((out_row >= 0) && (out_row < height ) && (out_col >= 0) && (out_col < width ) ) {
                sum += input_tile[threadIdx.y + filter_row - FILTER_RADIUS][threadIdx.x + filter_col - FILTER_RADIUS] * filter_c[filter_row][filter_col];
            }
        }
    }
    __syncthreads();

}

void copyFilterToGPU(float filter[][FILTER_DIM]) {
    // Copy filter to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(filter_c), filter, FILTER_DIM*FILTER_DIM*sizeof(float));
}

void convolution_tiled_gpu(float* input_d, float* output_d, unsigned int width, unsigned int height) {
    
    /*enough threads per block to load an entire input tile,
      enough blocks in the grid to process every output tile */

    dim3 numThreadsPerBlock(IN_TILE_DIM, IN_TILE_DIM);
    dim3 numBlocks((width + IN_TILE_DIM - 1)/IN_TILE_DIM, (height + IN_TILE_DIM - 1)/IN_TILE_DIM);
    convolution_tiled_kernel <<< numBlocks, numThreadsPerBlock >>> (input_d, output_d, width, height);
    hipDeviceSynchronize(); 
}

